#include "hip/hip_runtime.h"
#include <stdio.h>

__device__
int get_global_id() {
	return blockDim.x*blockIdx.x + threadIdx.x;
}
  
__global__ void cuda_hello()
{
  /* TODO: find global thread id */
  int tid = get_global_id();
  printf("Hello World from thread %d!\n", tid);
}

#define BLOCK_DIM 1
#define GRID_DIM 64

int main() {
  /* TODO: define thread block and grid dimensions */
  dim3 block(BLOCK_DIM);
  dim3 grid(GRID_DIM);
  cuda_hello<<<grid, block>>>(); 

  # this call is for cpu to wait gpu excecution
  hipDeviceSynchronize();
    
  return 0;
}
