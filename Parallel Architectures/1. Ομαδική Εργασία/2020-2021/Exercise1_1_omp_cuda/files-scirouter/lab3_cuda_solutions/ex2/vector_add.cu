#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

char *program_name = NULL;

double csecond(void) {
  struct timespec tms;

  if (clock_gettime(CLOCK_REALTIME, &tms)) {
    return (0.0);
  }
  /// seconds, multiplied with 1 million
  int64_t micros = tms.tv_sec * 1000000;
  /// Add full microseconds
  micros += tms.tv_nsec / 1000;
  /// round up if necessary
  if (tms.tv_nsec % 1000 >= 500) {
    ++micros;
  }
  return ((double)micros / 1000000.0);
}

static void check(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result) {
    printf("CUDA error at %s: %d code = %d (%s) %s\n", file, line, static_cast<unsigned int>(result), hipGetErrorName(result), func);
    exit(EXIT_FAILURE);
  }
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

static void set_program_name(char *path) {
  if (!program_name)
    program_name = strdup(path);
  if (!program_name)
    fprintf(stderr, "strdup failed\n");
}

static void print_usage() {
  printf("Usage: %s <N>\n", program_name);
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
  for (int i = 0; i < n; i++) {
    out[i] = a[i] + b[i];
  }
}

int main(int argc, char **argv) {
  set_program_name(argv[0]);
  if (argc < 2) {
    printf("Error in number of arguments!\n");
    print_usage();
    exit(1);
  }

  int N = atoi(argv[1]);
  float *a, *b, *out; 

  // Allocate memory for vectors
  a   = (float*)malloc(sizeof(float) * N);
  b   = (float*)malloc(sizeof(float) * N);
  out = (float*)malloc(sizeof(float) * N);

  // Initialize vectors
  for (int i = 0; i < N; i++) {
    a[i] = 1.0f; b[i] = 2.0f;
  }

  double timer = csecond();

  dim3 block(1);
  dim3 grid(1);
  vector_add<<<grid, block>>>(out, a, b, N);
  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipDeviceSynchronize());

  timer = csecond() - timer;
  printf("Addition completed in %lf seconds!\n", timer);
  
  // Cleanup
  free(a);
  free(b);
  free(out);

  return 0;
}
