#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#define EPS 1e-4

char *program_name = NULL;

double csecond(void) {
  struct timespec tms;

  if (clock_gettime(CLOCK_REALTIME, &tms)) {
    return (0.0);
  }
  /// seconds, multiplied with 1 million
  int64_t micros = tms.tv_sec * 1000000;
  /// Add full microseconds
  micros += tms.tv_nsec / 1000;
  /// round up if necessary
  if (tms.tv_nsec % 1000 >= 500) {
    ++micros;
  }
  return ((double)micros / 1000000.0);
}

static void set_program_name(char *path) {
  if (!program_name)
    program_name = strdup(path);
  if (!program_name)
    fprintf(stderr, "strdup failed\n");
}

static void print_usage() {
  printf("Usage: %s <N>\n", program_name);
}

static void check(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result) {
    printf("CUDA error at %s: %d code = %d (%s) %s\n", file, line, static_cast<unsigned int>(result), hipGetErrorName(result), func);
    exit(EXIT_FAILURE);
  }
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__device__ 
int get_global_id(){
  return blockDim.x*blockIdx.x + threadIdx.x;
}

static void dot_product_cpu(float *out, float *a, float *b, int n) {
  for (int i = 0; i < n; i++) {
    *out += a[i] * b[i];
  }
}

/* TODO: optimize the GPU implementation */
__global__ void dot_product_gpu(float *out, float *a, float *b, int n) {
  for (int i = 0; i < n; i++) {
    *out += a[i] * b[i];
  }
}

int main(int argc, char **argv) {
  set_program_name(argv[0]);
  if (argc < 2) {
    printf("Error in number of arguments!\n");
    print_usage();
    exit(1);
  }

  int N = atoi(argv[1]);
  float *a, *b, out = 0.;
  float *dev_a, *dev_b, *dev_out;
	
  // Allocate memory on the CPU memory
  a = (float*)malloc(N * sizeof(float));
  b = (float*)malloc(N * sizeof(float));
	
  // Allocate memory on the GPU memory
  checkCudaErrors(hipMalloc((void**)&dev_a, N*sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dev_b, N*sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dev_out, sizeof(float)));
	
  // Initialize vectors
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = i*2;
  }

  // Copy the vectors to the GPU
  checkCudaErrors(hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev_out, &out, sizeof(float), hipMemcpyHostToDevice));

  double timer = csecond();

  /* TODO: optimize the grid and block dimensions */
  dim3 grid(1);
  dim3 block(1);  
  dot_product_gpu<<<grid, block>>>(dev_out, dev_a, dev_b, N);
  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipDeviceSynchronize());

  timer = csecond() - timer;
  printf("GPU Product completed in %lf seconds!\n", timer);

  // Copy the result back from the GPU to the CPU
  checkCudaErrors(hipMemcpy(&out, dev_out, sizeof(float), hipMemcpyDeviceToHost));


  // Check result
  float out_test = 0;

  timer = csecond();
  dot_product_cpu(&out_test, a, b, N);
  timer = csecond() - timer;
  printf("CPU Product completed in %lf seconds!\n", timer);

  if (fabs((float)(out - out_test) / (float)out) > EPS) {
    printf("Result differs: %lf vs %lf\n", out, out_test);
    printf("Test FAILED!\n");
  } else {
    printf("Test PASSED!\n");
  }
    
  // Free memory on the GPU side
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_out);
	
  // Free memory on the CPU side
  free(a);
  free(b);
}
