
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ 
int get_global_id(){
  return blockDim.x*blockIdx.x + threadIdx.x;
}

__global__ void cuda_hello()
{
  int tid = get_global_id();
  printf("Hello World from thread %d!\n", tid);
}

#define BLOCK_DIM 1
#define GRID_DIM 64

int main() {

  dim3 block(BLOCK_DIM);
  dim3 grid(GRID_DIM);
  cuda_hello<<<grid, block>>>(); 
  hipDeviceSynchronize();
    
  return 0;
}
